#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <stdio.h>
#include "C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v12.2\include\hip/hip_runtime.h"
#include "C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v12.2\include\"
#include "../slib/mandelbrot.cuh"

float rmax = 1.5f;
float rmin = -1.5f;
            // make sure the aspect ratio is 16:9
float imin = -1.5f * 9.0f / 16.0f;
float imax = 1.5f * 9.0f / 16.0f;

int max_iterations = 100;

// CUDA Kernel
__global__ void mandelbrotKernel(int* output, int width, int height, float rmin, float rmax, float imin, float imax, int max_iterations) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x >= width || y >= height) return;

    double x0 = (double)x / (double)width * (rmax - rmin) + rmin;
    double y0 = (double)y / (double)height * (imax - imin) + imin;
    hipDoubleComplex z0 = make_hipDoubleComplex(x0, y0);
    hipDoubleComplex z = z0;
    int iterations = 0;

    while(hipCabs(z) < 2.0f && iterations < max_iterations) {
        z = hipCadd(hipCmul(z, z), z0);
        iterations++;
    }
    output[y * width + x] = iterations;
}

// Wrapper function to invoke the CUDA kernel
void computeMandelbrotGPU(int* h_output, int width, int height) {
    int* d_output;

    hipMalloc(&d_output, width * height * sizeof(int));

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    mandelbrotKernel<<<numBlocks, threadsPerBlock>>>(d_output, width, height, rmin, rmax, imin, imax, max_iterations);

    hipMemcpy(h_output, d_output, width * height * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_output);
}

int initializeCUDA() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if(deviceCount == 0) {
        return -1;
    }
    return 0;
}

void computeMandelbrot(int* pixels, int width, int height) {
    for (int i = 0; i < width * height; i++) {
        pixels[i] = i;
    }
    computeMandelbrotGPU(pixels, width, height); 
}

// Set new mandelbrot range
void set_mandelbrot_range(float new_rmin, float new_rmax, float new_imin, float new_imax) {
    rmin = new_rmin;
    rmax = new_rmax;
    imin = new_imin;
    imax = new_imax;
}

// map the number of iterations to a color
int color(int iterations) {
    int ratio = 0xFFFFFF / max_iterations;
    int color = iterations * ratio;
    return color;
}
